#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

template <typename T>
__global__ void sgd_reference(bool is_first_time, int numel, T* param, T* grad,
                              T* momentum_buffer, T lr, T weight_decay,
                              T momentum, T dampening, bool nesterov,
                              bool maximize) {
  int xid = blockIdx.x * blockDim.x + threadIdx.x;
  if (xid >= numel) return;

  T g = maximize ? -grad[xid] : grad[xid];
  g += weight_decay * param[xid];
  momentum_buffer[xid] =
      is_first_time
          ? g
          : (momentum * momentum_buffer[xid] + ((T)1 - dampening) * g);
  g = nesterov ? (g + momentum * momentum_buffer[xid]) : momentum_buffer[xid];
  param[xid] -= g * lr;
}

extern "C" __global__ void sgd_reference_fp32(bool is_first_time, int numel,
                                              float* param, float* grad,
                                              float* momentum_buffer, float lr,
                                              float weight_decay,
                                              float momentum, float dampening,
                                              bool nesterov, bool maximize) {
  sgd_reference(is_first_time, numel, param, grad, momentum_buffer, lr,
                weight_decay, momentum, dampening, nesterov, maximize);
}

extern "C" __global__ void sgd_reference_fp16(bool is_first_time, int numel,
                                              half* param, half* grad,
                                              half* momentum_buffer, half lr,
                                              half weight_decay, half momentum,
                                              half dampening, bool nesterov,
                                              bool maximize) {
  sgd_reference(is_first_time, numel, param, grad, momentum_buffer, lr,
                weight_decay, momentum, dampening, nesterov, maximize);
}